#include "hip/hip_runtime.h"
#include "FiringRateModel_Cuda.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <iostream>

#define Threads_Per_Block 512
#define MAX_SYNAPSES_PER_THREAD 256
#define MAX_NEURONS Threads_Per_Block

#define xcuda(stmt) {                                                   \
        hipError_t err = stmt;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << __FILE__ << ":" << __LINE__ << ": Failed to run " << #stmt << ". Reason: " << hipGetErrorString(err) << std::endl; \
            assert(false);                                                    \
        }                                                               \
    }

FiringRateModel_Cuda::FiringRateModel_Cuda() {
    memset(&gpu.buffers, 0, sizeof(gpu.buffers));
}

FiringRateModel_Cuda::~FiringRateModel_Cuda() {
#define FREE(x) xcuda( hipFree(gpu.buffers.x) );

    FREE(neurons);
    FREE(synapses);
    FREE(partitions);
    FREE(neuronactivation);
    FREE(newneuronactivation);
    FREE(efficacy);
}

void FiringRateModel_Cuda::init(FiringRateModel__Neuron *neurons,
                                short neurons_count, short input_neurons_count, short output_neurons_count,
                                float *neuronactivation,
                                FiringRateModel__Synapse *synapses,
                                long synapses_count,
                                float logistic_slope,
                                float decay_rate,
                                float max_weight) {

    assert(neurons_count < MAX_NEURONS);

    gpu.neurons_count = neurons_count;
    gpu.input_neurons_count = input_neurons_count;
    gpu.output_neurons_count = output_neurons_count;
    gpu.synapses_count = synapses_count;
    gpu.logistic_slope = logistic_slope;
    gpu.decay_rate = decay_rate;
    gpu.max_weight = max_weight;

    int nsynapses_per_thread = (synapses_count - 1) / Threads_Per_Block + 1;
    assert(nsynapses_per_thread <= MAX_SYNAPSES_PER_THREAD);

    Neuron gpu_neurons[neurons_count];
    for(short i = 0; i < neurons_count; i++) {
        gpu_neurons[i].bias = neurons[i].bias;
        gpu_neurons[i].tau = neurons[i].tau;
    }

    NeuronActivationPartition partitions[USHRT_MAX];
    NeuronActivationPartition *currpartition = NULL;
    Synapse gpu_synapses[synapses_count];
    float efficacy[synapses_count];

    for(long i = 0; i < synapses_count; i++) {
        FiringRateModel__Synapse &synapse = synapses[i];
        if( (i % Threads_Per_Block == 0) || (synapse.toneuron != currpartition->toneuron) ) {
            if(currpartition)
                currpartition++;
            else
                currpartition = partitions;
            assert(currpartition - partitions < USHRT_MAX);

            currpartition->toneuron = synapse.toneuron;
            currpartition->offset = i % Threads_Per_Block;
            currpartition->len = 0;
        }
        currpartition->len++;

        Synapse &gpu_synapse = gpu_synapses[i];
        gpu_synapse.fromneuron = synapse.fromneuron;
        gpu_synapse.partition = currpartition - partitions;
        gpu_synapse.lrate = synapse.lrate;

        efficacy[i] = synapse.efficacy;
    }

    size_t npartitions = currpartition - partitions + 1;
    gpu.partitions_count = npartitions;

/*
    for(long i = 0; i < synapses_count; i++) {
        if( (i % Threads_Per_Block) == 0 ) {
            printf("********\n");
        }
        NeuronActivationPartition &p = partitions[partition_index[i]];
        assert(p.toneuron == synapses[i].toneuron);
        printf("%5ld %3d] %3d %3d %3d\n", i, synapses[i].toneuron, p.toneuron, p.offset, p.len);
    }
    for(size_t i = 0; i < npartitions; i++) {
        NeuronActivationPartition &p = partitions[i];
        printf("%4lu] to=%3d off=%4d len=%3d\n", i, p.toneuron, p.offset, p.len);
    }
*/

    xcuda( hipMalloc((void**)&gpu.buffers.neurons, sizeof(gpu_neurons)) );
    xcuda( hipMemcpy(gpu.buffers.neurons, gpu_neurons, sizeof(gpu_neurons), hipMemcpyHostToDevice) );

    xcuda( hipMalloc((void**)&gpu.buffers.synapses, sizeof(gpu_synapses)) );
    xcuda( hipMemcpy(gpu.buffers.synapses, gpu_synapses, sizeof(gpu_synapses), hipMemcpyHostToDevice) );

    size_t sizeof_partitions = npartitions * sizeof(NeuronActivationPartition);
    xcuda( hipMalloc((void**)&gpu.buffers.partitions, sizeof_partitions) );
    xcuda( hipMemcpy(gpu.buffers.partitions, partitions, sizeof_partitions, hipMemcpyHostToDevice) );

    gpu.buffers.inputactivation = NULL;

    size_t sizeof_activation = sizeof(float) * neurons_count;
    xcuda( hipMalloc((void **)&gpu.buffers.neuronactivation, sizeof_activation) );
    xcuda( hipMemcpy(gpu.buffers.neuronactivation, neuronactivation, sizeof_activation, hipMemcpyHostToDevice) );

    xcuda( hipMalloc((void **)&gpu.buffers.newneuronactivation, sizeof_activation) );

    xcuda( hipMalloc((void**)&gpu.buffers.efficacy, sizeof(efficacy)) );
    xcuda( hipMemcpy(gpu.buffers.efficacy, efficacy, sizeof(efficacy), hipMemcpyHostToDevice) );
}

__device__ void sum_partition(float *x, int i, int n, float *result) {
    int stride = __popc(n) == 1 ? n >> 1 : 1 << 31 - __clz(n);

    if(i + stride < n) {
        x[i] += x[i + stride];
    }
      
    __syncthreads();

    stride >>= 1;
    // max_stride necessary to keep all threads from all partitions in sync.
    for(int max_stride = Threads_Per_Block >> 4; max_stride > 0; stride >>= 1, max_stride >>= 1) {
        if(i < stride) {
            x[i] += x[i + stride];
        }
        __syncthreads();
    }

    if(i == 0) {
        *result += x[0];
    }

    __syncthreads();
}

static __device__ float logistic(float x, float slope) {
    return (1.0 / (1.0 + exp(-1 * x * slope)));
}

__global__ void update(FiringRateModel_Cuda::GpuState *states) {
    int tid = threadIdx.x;

    FiringRateModel_Cuda::GpuState state = states[blockIdx.x];

    extern __shared__ char __shared_buf[];

    float *neuronactivation = (float *)__shared_buf;
    float *newneuronactivation = neuronactivation + state.neurons_count;
    float *partial_activation = newneuronactivation + state.neurons_count;

    if(tid < state.input_neurons_count) {
        state.buffers.neuronactivation[tid] = state.buffers.inputactivation[tid];
    }

    FiringRateModel_Cuda::Neuron neuron;
    if(tid < state.neurons_count) {
        neuron = state.buffers.neurons[tid];
        neuronactivation[tid] = state.buffers.neuronactivation[tid];
        newneuronactivation[tid] = neuron.bias;
    }
    __syncthreads();

    FiringRateModel_Cuda::Synapse synapses[MAX_SYNAPSES_PER_THREAD];
    float efficacies[MAX_SYNAPSES_PER_THREAD];
    const int nits = 1 + (state.synapses_count - 1) / Threads_Per_Block;

    for(int i = tid, it = 0; it < nits; i += Threads_Per_Block, it++) {
        if(i < state.synapses_count) {
            synapses[it] = state.buffers.synapses[i];
            efficacies[it] = state.buffers.efficacy[i];
            partial_activation[tid] = efficacies[it] * neuronactivation[synapses[it].fromneuron];
        }
        __syncthreads();

        float *partition_x;
        int partition_i;
        int partition_n;
        float *result;
        
        if(i < state.synapses_count) {
            FiringRateModel_Cuda::NeuronActivationPartition p = state.buffers.partitions[synapses[it].partition];

            partition_x = partial_activation + p.offset;
            partition_i = tid - p.offset;
            partition_n = p.len;
            result = newneuronactivation + p.toneuron;
        } else {
            partition_x = NULL;
            partition_i = 1;
            partition_n = 0;
            result = NULL;
        }

        sum_partition(partition_x,
                      partition_i,
                      partition_n,
                      result);

        __syncthreads();
    }

    if(tid < state.neurons_count) {
        newneuronactivation[tid] =
            (1.0f - neuron.tau) * neuronactivation[tid]
            + neuron.tau * logistic( newneuronactivation[tid], state.logistic_slope );
    }
    __syncthreads();

    for(int i = tid; i < state.synapses_count; i += Threads_Per_Block) {
        FiringRateModel_Cuda::Synapse synapse = state.buffers.synapses[i];
        short toneuron = state.buffers.partitions[synapse.partition].toneuron;
        float efficacy = state.buffers.efficacy[i];

        efficacy += synapse.lrate
            * (newneuronactivation[toneuron] - 0.5f)
            * (neuronactivation[synapse.fromneuron] - 0.5f);

        if (abs(efficacy) > (0.5f * state.max_weight)) {
            efficacy *= 1.0f - (1.0f - state.decay_rate) *
                (abs(efficacy) - 0.5f * state.max_weight) / (0.5f * state.max_weight);
            if (efficacy > state.max_weight)
                efficacy = state.max_weight;
            else if (efficacy < -state.max_weight)
                efficacy = -state.max_weight;
        } else {
            // not strictly correct for this to be in an else clause,
            // but if lrate is reasonable, efficacy should never change
            // sign with a new magnitude greater than 0.5 * Brain::config.maxWeight
            if (synapse.lrate >= 0.0f)  // excitatory
                efficacy = max(0.0f, efficacy);
            if (synapse.lrate < 0.0f)  // inhibitory
                efficacy = min(-1.e-10f, efficacy);
        }

        state.buffers.efficacy[i] = efficacy;
    }

    for(int i = tid; i < state.neurons_count; i += Threads_Per_Block) {
        state.buffers.newneuronactivation[i] = newneuronactivation[i];
    }
}

void FiringRateModel_Cuda::update(float *neuronactivation,
                                  float *newneuronactivation,
                                  FiringRateModel__Synapse *synapses) {

    xcuda( hipMalloc((void**)&gpu.buffers.inputactivation, sizeof(float) * gpu.input_neurons_count) );
    xcuda( hipMemcpy(gpu.buffers.inputactivation, neuronactivation, sizeof(float)*gpu.input_neurons_count, hipMemcpyHostToDevice) );

    assert(false);
    //size_t sizeof_shared = (2 * gpu.neurons_count + Threads_Per_Block) * sizeof(float);
    //::update<<<1, Threads_Per_Block, sizeof_shared>>>( gpu );

    xcuda( hipFree(gpu.buffers.inputactivation) );
    gpu.buffers.inputactivation = NULL;

#if !EXEC_CPU
    // todo: why do we need to copy the input neurons as well?
    xcuda( hipMemcpy(newneuronactivation,
                      gpu.buffers.newneuronactivation,
                      sizeof(float) * (gpu.output_neurons_count+gpu.input_neurons_count),
                      hipMemcpyDeviceToHost) );

/*
    xcuda( hipMemcpy(newneuronactivation + gpu.input_neurons_count,
                      gpu.buffers.newneuronactivation + gpu.input_neurons_count,
                      sizeof(float) * gpu.output_neurons_count, hipMemcpyDeviceToHost) );
*/
#else
    static int it = -1;
    it++;
    
    bool is_error = false;

    float test_activation[gpu.neurons_count];
    xcuda( hipMemcpy(test_activation, gpu.buffers.newneuronactivation, sizeof(test_activation), hipMemcpyDeviceToHost) );
    for(int i = gpu.input_neurons_count; i < gpu.neurons_count; i++) {
        float expected = newneuronactivation[i];
        float actual = test_activation[i];
        float error = fabs(actual - expected);
        if(error > 0.20) {
            std::cerr << "bad neuron " << i << ": expected=" << newneuronactivation[i] << ", actual=" << test_activation[i] << ", error=" << error << std::endl;
            is_error = true;
            break;
        }
    }
    for(int i = 0; i < gpu.neurons_count; i++) {
        newneuronactivation[i] = test_activation[i];
    }

    float test_efficacy[gpu.synapses_count];
    xcuda( hipMemcpy(test_efficacy, gpu.buffers.efficacy, sizeof(test_efficacy), hipMemcpyDeviceToHost) );
    for(int i = 0; i < gpu.synapses_count; i++) {
        float expected = synapses[i].efficacy;
        float actual = test_efficacy[i];
        float error = fabs(actual - expected);
        if(error > 0.01) {
            std::cerr << "bad synapse " << i << ": expected=" << expected << ", actual=" << actual << ", error=" << error << std::endl;
            is_error = true;
            break;
        }
        synapses[i].efficacy = test_efficacy[i];
    }

    if(is_error) {
        std::cerr << "it=" << it << std::endl;
        exit(0);
    }
#endif

    {
        float *swap = gpu.buffers.neuronactivation;
        gpu.buffers.neuronactivation = gpu.buffers.newneuronactivation;
        gpu.buffers.newneuronactivation = swap;
    }
}

void FiringRateModel_Cuda::update(AgentState *agents, long nagents) {
    GpuState gpus[nagents];

    for(long i = 0; i < nagents; i++) {
        AgentState &agent = agents[i];
        GpuState *gpu = &agent.model->gpu;

        xcuda( hipMalloc((void**)&gpu->buffers.inputactivation, sizeof(float) * gpu->input_neurons_count) );
        xcuda( hipMemcpy(gpu->buffers.inputactivation, agent.neuronactivation, sizeof(float)*gpu->input_neurons_count, hipMemcpyHostToDevice) );

        gpus[i] = *gpu;
    }

    GpuState *d_gpus;
    xcuda( hipMalloc((void**)&d_gpus, sizeof(gpus)) );
    xcuda( hipMemcpy(d_gpus, gpus, sizeof(gpus), hipMemcpyHostToDevice) );

    uint sizeof_shared = 0;
    for(long i = 0; i < nagents; i++) {
        GpuState &gpu = gpus[i];
        sizeof_shared = max(sizeof_shared, uint((2 * gpu.neurons_count + Threads_Per_Block) * sizeof(float)));
    }

    ::update<<<nagents, Threads_Per_Block, sizeof_shared>>>(d_gpus);

    for(long i = 0; i < nagents; i++) {
        AgentState &agent = agents[i];
        GpuState &gpu = gpus[i];

        // todo: why do we need to copy the input neurons as well?
        xcuda( hipMemcpy(agent.newneuronactivation,
                          gpu.buffers.newneuronactivation,
                          sizeof(float) * (gpu.output_neurons_count+gpu.input_neurons_count),
                          hipMemcpyDeviceToHost) );
    }

    for(long i = 0; i < nagents; i++) {
        GpuState &gpu = gpus[i];

        xcuda( hipFree(gpu.buffers.inputactivation) );
    }
    
    xcuda( hipFree(d_gpus) );

    for(long i = 0; i < nagents; i++) {
        GpuState *gpu = &agents[i].model->gpu;
        float *swap = gpu->buffers.neuronactivation;
        gpu->buffers.neuronactivation = gpu->buffers.newneuronactivation;
        gpu->buffers.newneuronactivation = swap;
    }
    
}
