#include "hip/hip_runtime.h"
#include "FiringRateModel_Cuda.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <iostream>

#define Threads_Per_Block 512
#define MAX_SYNAPSES_PER_THREAD 256
#define MAX_NEURONS Threads_Per_Block

#define xcuda(stmt) {                                                   \
        hipError_t err = stmt;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << __FILE__ << ":" << __LINE__ << ": Failed to run " << #stmt << ". Reason: " << hipGetErrorString(err) << std::endl; \
            assert(false);                                                    \
        }                                                               \
    }

FiringRateModel_Cuda::FiringRateModel_Cuda() {
    memset(&gpu.buffers, 0, sizeof(gpu.buffers));
}

FiringRateModel_Cuda::~FiringRateModel_Cuda() {
    xcuda( hipHostFree(buffer) );
    xcuda( hipFree(d_buffer) );
}

void FiringRateModel_Cuda::init(FiringRateModel__Neuron *neurons,
                                short neurons_count, short input_neurons_count, short output_neurons_count,
                                float *neuronactivation,
                                FiringRateModel__Synapse *synapses,
                                long synapses_count,
                                float logistic_slope,
                                float decay_rate,
                                float max_weight) {

    assert(neurons_count < MAX_NEURONS);

    gpu.neurons_count = neurons_count;
    gpu.input_neurons_count = input_neurons_count;
    gpu.output_neurons_count = output_neurons_count;
    gpu.synapses_count = synapses_count;
    gpu.logistic_slope = logistic_slope;
    gpu.decay_rate = decay_rate;
    gpu.max_weight = max_weight;

    int nsynapses_per_thread = (synapses_count - 1) / Threads_Per_Block + 1;
    assert(nsynapses_per_thread <= MAX_SYNAPSES_PER_THREAD);

    NeuronActivationPartition partitions[USHRT_MAX];
    size_t partitions_count = 0;
    Synapse gpu_synapses[synapses_count];
    float efficacy[synapses_count];
    {
        NeuronActivationPartition *currpartition = NULL;

        for(long i = 0; i < synapses_count; i++) {
            FiringRateModel__Synapse &synapse = synapses[i];
            if( (i % Threads_Per_Block == 0) || (synapse.toneuron != currpartition->toneuron) ) {
                if(currpartition)
                    currpartition++;
                else
                    currpartition = partitions;
                assert(currpartition - partitions < USHRT_MAX);

                currpartition->toneuron = synapse.toneuron;
                currpartition->offset = i % Threads_Per_Block;
                currpartition->len = 0;
            }
            currpartition->len++;

            Synapse &gpu_synapse = gpu_synapses[i];
            gpu_synapse.fromneuron = synapse.fromneuron;
            gpu_synapse.partition = currpartition - partitions;
            gpu_synapse.lrate = synapse.lrate;

            efficacy[i] = synapse.efficacy;
        }

        partitions_count = currpartition - partitions + 1;
        gpu.partitions_count = partitions_count;
    }

    uint sizeof_neurons = neurons_count * sizeof(Neuron);
    uint sizeof_synapses = synapses_count * sizeof(Synapse);
    size_t sizeof_partitions = partitions_count * sizeof(NeuronActivationPartition);
    size_t sizeof_activation = neurons_count * sizeof(float);
    size_t sizeof_efficacy = synapses_count * sizeof(float);

    size_t sizeof_buffer =
        sizeof_neurons
        + sizeof_synapses
        + sizeof_partitions
        + sizeof_activation
        + sizeof_efficacy;

    xcuda( hipHostMalloc((void **)&buffer, sizeof_buffer) );
    xcuda( hipMalloc((void **)&d_buffer, sizeof_buffer) );

    {
        uint offset = 0;
        {
            Neuron *gpu_neurons = (Neuron *)buffer + offset;
            for(short i = 0; i < neurons_count; i++) {
                gpu_neurons[i].bias = neurons[i].bias;
                gpu_neurons[i].tau = neurons[i].tau;
            }
        }
        gpu.buffers.neurons = (Neuron *)(d_buffer + offset);
        offset += sizeof_neurons;

        memcpy(buffer + offset, gpu_synapses, sizeof_synapses);
        gpu.buffers.synapses = (Synapse *)(d_buffer + offset);
        offset += sizeof_synapses;

        memcpy(buffer + offset, partitions, sizeof_partitions);
        gpu.buffers.partitions = (NeuronActivationPartition *)(d_buffer + offset);
        offset += sizeof_partitions;

        memcpy(buffer + offset, neuronactivation, sizeof_activation);
        gpu.buffers.neuronactivation = (float *)(d_buffer + offset);
        offset += sizeof_activation;

        memcpy(buffer + offset, efficacy, sizeof_efficacy);
        gpu.buffers.efficacy = (float *)(d_buffer + offset);
        offset += sizeof_efficacy;
    }

    xcuda( hipMemcpy(d_buffer, buffer, sizeof_buffer, hipMemcpyHostToDevice) );
}

__device__ void sum_partition(float *x, int i, int n, float *result) {
    int stride = __popc(n) == 1 ? n >> 1 : 1 << 31 - __clz(n);

    if(i + stride < n) {
        x[i] += x[i + stride];
    }
      
    __syncthreads();

    stride >>= 1;
    // max_stride necessary to keep all threads from all partitions in sync.
    for(int max_stride = Threads_Per_Block >> 4; max_stride > 0; stride >>= 1, max_stride >>= 1) {
        if(i < stride) {
            x[i] += x[i + stride];
        }
        __syncthreads();
    }

    if(i == 0) {
        *result += x[0];
    }

    __syncthreads();
}

static __device__ float logistic(float x, float slope) {
    return (1.0 / (1.0 + exp(-1 * x * slope)));
}

__global__ void update(FiringRateModel_Cuda::GpuState *states) {
    int tid = threadIdx.x;

    FiringRateModel_Cuda::GpuState state = states[blockIdx.x];

    extern __shared__ char __shared_buf[];

    float *neuronactivation = (float *)__shared_buf;
    float *newneuronactivation = neuronactivation + state.neurons_count;
    float *partial_activation = newneuronactivation + state.neurons_count;

    FiringRateModel_Cuda::Neuron neuron;
    if(tid < state.neurons_count) {
        neuron = state.buffers.neurons[tid];        
    	if(tid < state.input_neurons_count) {
        	neuronactivation[tid] = state.buffers.input_activation[tid];
            newneuronactivation[tid] = neuronactivation[tid];
    	} else {
			neuronactivation[tid] = state.buffers.neuronactivation[tid];
            newneuronactivation[tid] = neuron.bias;
		}
    }
    __syncthreads();

    FiringRateModel_Cuda::Synapse synapses[MAX_SYNAPSES_PER_THREAD];
    float efficacies[MAX_SYNAPSES_PER_THREAD];
    const int nits = 1 + (state.synapses_count - 1) / Threads_Per_Block;

    for(int i = tid, it = 0; it < nits; i += Threads_Per_Block, it++) {
        if(i < state.synapses_count) {
            synapses[it] = state.buffers.synapses[i];
            efficacies[it] = state.buffers.efficacy[i];
            partial_activation[tid] = efficacies[it] * neuronactivation[synapses[it].fromneuron];
        }
        __syncthreads();

        float *partition_x;
        int partition_i;
        int partition_n;
        float *result;
        
        if(i < state.synapses_count) {
            FiringRateModel_Cuda::NeuronActivationPartition p = state.buffers.partitions[synapses[it].partition];

            partition_x = partial_activation + p.offset;
            partition_i = tid - p.offset;
            partition_n = p.len;
            result = newneuronactivation + p.toneuron;
        } else {
            partition_x = NULL;
            partition_i = 1;
            partition_n = 0;
            result = NULL;
        }

        sum_partition(partition_x,
                      partition_i,
                      partition_n,
                      result);

        __syncthreads();
    }

    if( (tid >= state.input_neurons_count) && (tid < state.neurons_count) ) {
        newneuronactivation[tid] =
            (1.0f - neuron.tau) * neuronactivation[tid]
            + neuron.tau * logistic( newneuronactivation[tid], state.logistic_slope );
    }
    __syncthreads();

    for(int i = tid, it = 0; i < state.synapses_count; i += Threads_Per_Block, it++) {
        FiringRateModel_Cuda::Synapse synapse = synapses[it];
        short toneuron = state.buffers.partitions[synapse.partition].toneuron;
        float efficacy = efficacies[it];

        efficacy += synapse.lrate
            * (newneuronactivation[toneuron] - 0.5f)
            * (neuronactivation[synapse.fromneuron] - 0.5f);

        if (abs(efficacy) > (0.5f * state.max_weight)) {
            efficacy *= 1.0f - (1.0f - state.decay_rate) *
                (abs(efficacy) - 0.5f * state.max_weight) / (0.5f * state.max_weight);
            if (efficacy > state.max_weight)
                efficacy = state.max_weight;
            else if (efficacy < -state.max_weight)
                efficacy = -state.max_weight;
        } else {
            // not strictly correct for this to be in an else clause,
            // but if lrate is reasonable, efficacy should never change
            // sign with a new magnitude greater than 0.5 * Brain::config.maxWeight
            if (synapse.lrate >= 0.0f)  // excitatory
                efficacy = max(0.0f, efficacy);
            if (synapse.lrate < 0.0f)  // inhibitory
                efficacy = min(-1.e-10f, efficacy);
        }

        state.buffers.efficacy[i] = efficacy;
    }

    if(tid < state.neurons_count) {
        state.buffers.neuronactivation[tid] = newneuronactivation[tid];

        if( (tid >= state.input_neurons_count)
            && (tid < state.input_neurons_count + state.output_neurons_count) ) {
            state.buffers.output_activation[tid - state.input_neurons_count] = newneuronactivation[tid];
        }
    }
}

typedef FiringRateModel_Cuda::AgentState AgentState;
typedef FiringRateModel_Cuda::GpuState GpuState;

static GpuState *gpus = NULL;
static GpuState *d_gpus = NULL;
static uint sizeof_shared = 0;
static float *d_all_input = NULL;
static uint sizeof_input = 0;
static float *d_all_output = NULL;
static uint sizeof_output = 0;

void FiringRateModel_Cuda::alloc_update_buffers(AgentState *agents,
                                                long nagents,
                                                uint *input_offset,
                                                uint ninput,
                                                float **all_input,
                                                uint *output_offset,
                                                uint noutput,
                                                float **all_output) {
    if(d_all_input) {
        xcuda( hipHostFree(gpus) );
        xcuda( hipHostFree(*all_input) );
        xcuda( hipHostFree(*all_output) );
        xcuda( hipFree(d_gpus) );
        xcuda( hipFree(d_all_input) );
        xcuda( hipFree(d_all_output) );
    }

    sizeof_input = ninput * sizeof(float);
    sizeof_output = noutput * sizeof(float);
    uint sizeof_gpus = nagents * sizeof(GpuState);

    sizeof_shared = 0;
    for(long i = 0; i < nagents; i++) {
        AgentState &agent = agents[i];
        GpuState *gpu = &agent.model->gpu;
        uint sizeof_agent = uint((2 * gpu->neurons_count + Threads_Per_Block) * sizeof(float));
        sizeof_shared = max(sizeof_shared, sizeof_agent);
    }

    xcuda( hipHostMalloc(all_input, sizeof_input) );
    xcuda( hipHostMalloc(all_output, sizeof_output) );
    xcuda( hipHostMalloc((void **)&gpus, sizeof_gpus) );

    xcuda( hipMalloc((void**)&d_gpus, sizeof(GpuState) * nagents) );
    xcuda( hipMalloc((void**)&d_all_input, sizeof_input) );
    xcuda( hipMalloc((void**)&d_all_output, sizeof_output) );

    for(long i = 0; i < nagents; i++) {
        AgentState &agent = agents[i];
        GpuState *gpu = &agent.model->gpu;
        gpu->buffers.input_activation = d_all_input + input_offset[i];
        gpu->buffers.output_activation = d_all_output + output_offset[i];
    }

    for(long i = 0; i < nagents; i++) {
        gpus[i] = agents[i].model->gpu;
    }
    xcuda( hipMemcpy(d_gpus, gpus, sizeof_gpus, hipMemcpyHostToDevice) );
}

void FiringRateModel_Cuda::update_all(AgentState *agents,
                                      long nagents,
                                      float *all_input,
                                      float *all_output) {

    xcuda( hipMemcpy(d_all_input,
                      all_input,
                      sizeof_input,
                      hipMemcpyHostToDevice) );

    ::update<<<nagents, Threads_Per_Block, sizeof_shared>>>(d_gpus);

    xcuda( hipMemcpy(all_output,
                      d_all_output,
                      sizeof_output,
                      hipMemcpyDeviceToHost) );
}
